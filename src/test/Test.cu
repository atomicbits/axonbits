//
// Created by Peter Rigole on 2019-04-24.
//

#ifndef AXONBITS_TEST_H
#define AXONBITS_TEST_H

#include <hip/hip_runtime.h>
#include <string>


/**
 * Test base class.
 * Mind that classes with virtual functions can't have a header file.
 */
class Test {

public:

    Test() {}

    Test(const char* nameInit) : name(nameInit) {}

    ~Test() {}

    __host__
    virtual void test() {}

    __host__
    void checkCudaErrors() {
        hipDeviceSynchronize();
        hipError_t hipError_t;
        hipError_t = hipGetLastError();
        if(hipError_t != hipSuccess) {
            printf("%s device failure, cudaGetLastError() returned %d: %s\n", getName(), hipError_t, hipGetErrorString(hipError_t));
        } else {
            printf("%s device test successful\n", getName());
        }
    }

    __host__
    const char* getName() { return name; }

private:
    const char* name;

};


#endif //AXONBITS_TEST_H

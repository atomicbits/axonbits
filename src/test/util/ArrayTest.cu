#include "hip/hip_runtime.h"
//
// Created by Peter Rigole on 2019-04-19.
//

#include "ArrayTest.cuh"

/**
 * Must not be in a class, but in the global scope!
 */
__global__
void launchArrayTestDeviceTest(ArrayTest *test) {
    test->deviceTest();
    return;
}


__host__
ArrayTest::ArrayTest() : Test("ArrayTest") {
    arr = new Array<TestContainer>(5);
    arr->append(TestContainer(1.0, 2.0, 300.0, 1, 2, 100));
    arr->append(TestContainer(4.0, 5.0, 6.0, 4, 5, 6));
    arr->append(TestContainer(7.0, 8.0, 9.0, 7, 8, 9));

    hipDeviceSynchronize();
}

__host__
ArrayTest::~ArrayTest() {
    delete arr;
}

__host__
void ArrayTest::test() {
    launchArrayTestDeviceTest<<< 1, 1 >>>(this);
    checkCudaErrors();
    hostTest();
    printf("%s host test successful\n", getName());
}

__device__
void ArrayTest::deviceTest() {

    float ab = (*arr)[0].addAB();
    assert(ab == 3.0);
    (*arr)[0].setC(ab);

    int xy = (*arr)[2].addXY();
    assert(xy == 15); // should be 15!
    (*arr)[2].setZ(xy);

    float sum = 0;
    for(Array<TestContainer>::iterator i = arr->begin(); i != arr->end(); i++) {
        TestContainer &testContainer = *i;
        sum += testContainer.getC();
    }
    // 3.0 + 6.0 + 9.0 = 18.0 (remember that (*arr)[0]->setC(3.0) above)
    assert(sum  == 18);

    return;
}

__host__
void ArrayTest::hostTest() {
    assert((*arr)[0].getC() == 3.0); // should be 3.0!
    assert((*arr)[2].getZ() == 15); // should be 15!
    return;
}


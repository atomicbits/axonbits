//
// Created by Peter Rigole on 2019-03-08.
//

#include "Managed.cuh"

__host__
void *Managed::operator new(size_t len) {
    void *ptr;
    hipMallocManaged(&ptr, len);
    hipDeviceSynchronize();
    return ptr;
}

__host__
void Managed::operator delete(void *ptr) {
    hipDeviceSynchronize();
    hipFree(ptr);
}

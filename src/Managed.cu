//
// Created by Peter Rigole on 2019-03-08.
//

#include "Managed.cuh"

__host__
void *Managed::operator new(size_t len) {
    void *ptr;
    hipMallocManaged(&ptr, len);

    return ptr;
}

__host__
void Managed::operator delete(void *ptr) {

    hipFree(ptr);
}

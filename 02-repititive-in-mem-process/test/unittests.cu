#include "hip/hip_runtime.h"
//
// Created by Peter Rigole on 2019-04-19.
//

#include <hip/hip_runtime.h>

#include <stdio.h>
#include <unistd.h>
#include <signal.h>
#include <assert.h>

#include "util/ArrayTest.cu"


__global__
void run_device_test(ArrayTest *arrayTest) {
    arrayTest->deviceTest();
    return;
}

void run_host_test(ArrayTest *arrayTest) {
    arrayTest->hostTest();
    return;
}

void launch_test(ArrayTest *arrayTest) {

    run_device_test<<< 1, 1 >>>(arrayTest);
    hipDeviceSynchronize();

    hipError_t hipError_t;
    hipError_t = hipGetLastError();
    if(hipError_t != hipSuccess) {
        printf("Device failure, hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
        delete arrayTest;
        return;
    }

    printf("ArrayTest device test successful\n");

    run_host_test(arrayTest);

    printf("ArrayTest host test successful\n");

    delete arrayTest;
    return;
}


int main(int argc, char **argv) {

    launch_test(new ArrayTest());

    printf("Testing done.\n");

    hipDeviceReset();

    return 0;
}

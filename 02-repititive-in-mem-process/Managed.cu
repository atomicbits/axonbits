//
// Created by Peter Rigole on 2019-03-08.
//

#include "Managed.cuh"

void *Managed::operator new(size_t len) {
    void *ptr;
    hipMallocManaged(&ptr, len);
    hipDeviceSynchronize();
    return ptr;
}

void Managed::operator delete(void *ptr) {
    hipDeviceSynchronize();
    hipFree(ptr);
}
